#include "hip/hip_runtime.h"
/* $Id$
 *
 * This file is part of the ESPResSo distribution (http://www.espresso.mpg.de).
 * It is therefore subject to the ESPResSo license agreement which you
 * accepted upon receiving the distribution and by which you are
 * legally bound while utilizing this file in any form or way.
 * There is NO WARRANTY, not even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. 
 * You should have received a copy of that license along with this
 * program; if not, refer to http://www.espresso.mpg.de/license.html
 * where its current version can be found, or write to
 * Max-Planck-Institute for Polymer Research, Theory Group, 
 * PO Box 3148, 55021 Mainz, Germany. 
 * Copyright (c) 2002-2007; all rights reserved unless otherwise stated.
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

extern "C" {
#include "lbgpu.h"
}

#ifdef LB_GPU

/**defining structures residing in global memory */
/** struct for phys. values */
static LB_values_gpu *device_values = NULL;
/** structs for velocity densities */
static LB_nodes_gpu nodes_a;
static LB_nodes_gpu nodes_b;
/** struct for particle force */
static LB_particle_force_gpu *particle_force = NULL;
/** struct for particle position and veloctiy */
static LB_particle_gpu *particle_data = NULL;
/** struct for node force */
static LB_node_force_gpu node_f;
/** struct for storing particle rn seed */
static LB_particle_seed_gpu *part = NULL;

static LB_extern_nodeforce_gpu *extern_nodeforces = NULL;
/** pointer for bound index array*/
static int *boundindex;
/** pointers for additional cuda check flag*/
static int *gpu_check = NULL;
static int *h_gpu_check = NULL;

/** values for the kernel call */
static int threads_per_block;
static int blocks_per_grid;

/** values for the particle kernel */
static int threads_per_block_particles;
static int blocks_per_grid_particles;

/** values for the boundary init kernel */
static int threads_per_block_bound;
static int blocks_per_grid_bound;

static int threads_per_block_exf;
static int blocks_per_grid_exf;

static int threads_per_block_print;
static int blocks_per_grid_print;
static unsigned int intflag = 1;
/**defining size values for allocating global memory */
static size_t size_of_values;
static size_t size_of_forces;
static size_t size_of_positions;
static size_t size_of_seed;
static size_t size_of_boundindex;
static size_t size_of_extern_nodeforces;

/**parameters residing in constant memory */
static __constant__ LB_parameters_gpu para;
static __constant__ int number_of_bnodes;
static __constant__ float c_sound_sq = 1.f/3.f;

/**cudasteams for parallel computing on cpu and gpu */
hipStream_t stream[1];

hipError_t err;
/*-------------------------------------------------------*/
/*********************************************************/
/**device funktions call by kernel funktions */
/*********************************************************/
/*-------------------------------------------------------*/


/*-------------------------------------------------------*/

/** atomic add function for sveral cuda architectures */
/*@{
 * @param 
}*/
/*-------------------------------------------------------*/
__device__ inline void atomicadd(float* address, float value){
#if !defined __CUDA_ARCH__ || __CUDA_ARCH__ >= 200 // for Fermi, atomicAdd supports floats
  atomicAdd(address, value);
#elif __CUDA_ARCH__ >= 110
#warning Using slower atomicAdd emulation
// float-atomic-add from 
// [url="http://forums.nvidia.com/index.php?showtopic=158039&view=findpost&p=991561"]http://forums.nvidia.com/index.php?showtop...st&p=991561[/url]
  float old = value;
  while ((old = atomicExch(address, atomicExch(address, 0.0f)+old))!=0.0f);
#else
#error I need at least compute capability 1.1
#endif
}
/*-------------------------------------------------------*/
/**randomgenerator which generates numbers [0,1] */
/*@{
 * @param *rn	Pointer to randomnumber array of the local node or particle 
}*/
/*-------------------------------------------------------*/
__device__ void random_01(LB_randomnr_gpu *rn){

  const float mxi = 1.f/(float)(1ul<<31);
  unsigned int curr = rn->seed;

  curr = 1103515245 * curr + 12345;
  rn->randomnr[0] = (float)(curr & ((1ul<<31)-1))*mxi;
  curr = 1103515245 * curr + 12345;
  rn->randomnr[1] = (float)(curr & ((1ul<<31)-1))*mxi;
  rn->seed = curr;

}
/*-------------------------------------------------------*/
/** gaussian random nummber generator for thermalisation */
/*@{
 * @param *rn	Pointer to randomnumber array of the local node node or particle 
}*/
/*-------------------------------------------------------*/
__device__ void gaussian_random(LB_randomnr_gpu *rn){

  float x1, x2;
  float r2, fac;
  /* On every second call two gaussian random numbers are calculated
   via the Box-Muller transformation.*/
  /* draw two uniform random numbers in the unit circle */
  do {
    random_01(rn);
    x1 = 2.f*rn->randomnr[0]-1.f;
    x2 = 2.f*rn->randomnr[1]-1.f;
    r2 = x1*x1 + x2*x2;
  } while (r2 >= 1.f || r2 == 0.f);

  /* perform Box-Muller transformation */
  fac = sqrtf(-2.f*__logf(r2)/r2);
  rn->randomnr[0] = x2*fac;
  rn->randomnr[1] = x1*fac;
  
}
/*-------------------------------------------------------*/
/**tranformation from 1d array-index to xyz */
/*@{
 * @param index		node index / thread index (Input)
 * @param xyz		Pointer to calculated xyz array (Output)
 */
/*-------------------------------------------------------*/
__device__ void index_to_xyz(unsigned int index, unsigned int *xyz){

  xyz[0] = index%para.dim_x;
  index /= para.dim_x;
  xyz[1] = index%para.dim_y;
  index /= para.dim_y;
  xyz[2] = index;
}
/*-------------------------------------------------------*/
/**calculation of the modes from the velocitydensities (space-transform.)*/
/*@{
 * @param n_a		Pointer to local node residing in array a (Input)
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Output)
}*/
/*-------------------------------------------------------*/
__device__ void calc_m_from_n(LB_nodes_gpu n_a, unsigned int index, float *mode){

  /* mass mode */
  mode[0] = n_a.vd[0][index] + n_a.vd[1][index] + n_a.vd[2][index]
          + n_a.vd[3][index] + n_a.vd[4][index] + n_a.vd[5][index]
          + n_a.vd[6][index] + n_a.vd[7][index] + n_a.vd[8][index]
          + n_a.vd[9][index] + n_a.vd[10][index] + n_a.vd[11][index] + n_a.vd[12][index]
          + n_a.vd[13][index] + n_a.vd[14][index] + n_a.vd[15][index] + n_a.vd[16][index]
          + n_a.vd[17][index] + n_a.vd[18][index];

  /* momentum modes */
  mode[1] = (n_a.vd[1][index] - n_a.vd[2][index]) + (n_a.vd[7][index] - n_a.vd[8][index])
          + (n_a.vd[9][index] - n_a.vd[10][index]) + (n_a.vd[11][index] - n_a.vd[12][index])
          + (n_a.vd[13][index] - n_a.vd[14][index]);
  mode[2] = (n_a.vd[3][index] - n_a.vd[4][index]) + (n_a.vd[7][index] - n_a.vd[8][index])
          - (n_a.vd[9][index] - n_a.vd[10][index]) + (n_a.vd[15][index] - n_a.vd[16][index])
          + (n_a.vd[17][index] - n_a.vd[18][index]);
  mode[3] = (n_a.vd[5][index] - n_a.vd[6][index]) + (n_a.vd[11][index] - n_a.vd[12][index])
          - (n_a.vd[13][index] - n_a.vd[14][index]) + (n_a.vd[15][index] - n_a.vd[16][index])
          - (n_a.vd[17][index] - n_a.vd[18][index]);

  /* stress modes */
  mode[4] = -(n_a.vd[0][index]) + n_a.vd[7][index] + n_a.vd[8][index] + n_a.vd[9][index] + n_a.vd[10][index]
          + n_a.vd[11][index] + n_a.vd[12][index] + n_a.vd[13][index] + n_a.vd[14][index]
          + n_a.vd[15][index] + n_a.vd[16][index] + n_a.vd[17][index] + n_a.vd[18][index];
  mode[5] = n_a.vd[1][index] + n_a.vd[2][index] - (n_a.vd[3][index] + n_a.vd[4][index])
          + (n_a.vd[11][index] + n_a.vd[12][index]) + (n_a.vd[13][index] + n_a.vd[14][index])
          - (n_a.vd[15][index] + n_a.vd[16][index]) - (n_a.vd[17][index] + n_a.vd[18][index]);
  mode[6] = (n_a.vd[1][index] + n_a.vd[2][index]) + (n_a.vd[3][index] + n_a.vd[4][index])
          - (n_a.vd[11][index] + n_a.vd[12][index]) - (n_a.vd[13][index] + n_a.vd[14][index])
          - (n_a.vd[15][index] + n_a.vd[16][index]) - (n_a.vd[17][index] + n_a.vd[18][index])
          - 2.f*(n_a.vd[5][index] + n_a.vd[6][index] - (n_a.vd[7][index] + n_a.vd[8][index])
          - (n_a.vd[9][index] +n_a.vd[10][index]));
  mode[7] = n_a.vd[7][index] + n_a.vd[8][index] - (n_a.vd[9][index] + n_a.vd[10][index]);
  mode[8] = n_a.vd[11][index] + n_a.vd[12][index] - (n_a.vd[13][index] + n_a.vd[14][index]);
  mode[9] = n_a.vd[15][index] + n_a.vd[16][index] - (n_a.vd[17][index] + n_a.vd[18][index]);

  /* kinetic modes */
  mode[10] = -2.f*(n_a.vd[1][index] - n_a.vd[2][index]) + (n_a.vd[7][index] - n_a.vd[8][index])
           + (n_a.vd[9][index] - n_a.vd[10][index]) + (n_a.vd[11][index] - n_a.vd[12][index])
           + (n_a.vd[13][index] - n_a.vd[14][index]);
  mode[11] = -2.f*(n_a.vd[3][index] - n_a.vd[4][index]) + (n_a.vd[7][index] - n_a.vd[8][index])
           - (n_a.vd[9][index] - n_a.vd[10][index]) + (n_a.vd[15][index] - n_a.vd[16][index])
           + (n_a.vd[17][index] - n_a.vd[18][index]);
  mode[12] = -2.f*(n_a.vd[5][index] - n_a.vd[6][index]) + (n_a.vd[11][index] - n_a.vd[12][index])
           - (n_a.vd[13][index] - n_a.vd[14][index]) + (n_a.vd[15][index] - n_a.vd[16][index])
           - (n_a.vd[17][index] - n_a.vd[18][index]);
  mode[13] = (n_a.vd[7][index] - n_a.vd[8][index]) + (n_a.vd[9][index] - n_a.vd[10][index])
           - (n_a.vd[11][index] - n_a.vd[12][index]) - (n_a.vd[13][index] - n_a.vd[14][index]);
  mode[14] = (n_a.vd[7][index] - n_a.vd[8][index]) - (n_a.vd[9][index] - n_a.vd[10][index])
           - (n_a.vd[15][index] - n_a.vd[16][index]) - (n_a.vd[17][index] - n_a.vd[18][index]);
  mode[15] = (n_a.vd[11][index] - n_a.vd[12][index]) - (n_a.vd[13][index] - n_a.vd[14][index])
           - (n_a.vd[15][index] - n_a.vd[16][index]) + (n_a.vd[17][index] - n_a.vd[18][index]);
  mode[16] = n_a.vd[0][index] + n_a.vd[7][index] + n_a.vd[8][index] + n_a.vd[9][index] + n_a.vd[10][index]
           + n_a.vd[11][index] + n_a.vd[12][index] + n_a.vd[13][index] + n_a.vd[14][index]
           + n_a.vd[15][index] + n_a.vd[16][index] + n_a.vd[17][index] + n_a.vd[18][index]
           - 2.f*((n_a.vd[1][index] + n_a.vd[2][index]) + (n_a.vd[3][index] + n_a.vd[4][index])
           + (n_a.vd[5][index] + n_a.vd[6][index]));
  mode[17] = -(n_a.vd[1][index] + n_a.vd[2][index]) + (n_a.vd[3][index] + n_a.vd[4][index])
           + (n_a.vd[11][index] + n_a.vd[12][index]) + (n_a.vd[13][index] + n_a.vd[14][index])
           - (n_a.vd[15][index] + n_a.vd[16][index]) - (n_a.vd[17][index] + n_a.vd[18][index]);
  mode[18] = -(n_a.vd[1][index] + n_a.vd[2][index]) - (n_a.vd[3][index] + n_a.vd[4][index])
           - (n_a.vd[11][index] + n_a.vd[12][index]) - (n_a.vd[13][index] + n_a.vd[14][index])
           - (n_a.vd[15][index] + n_a.vd[16][index]) - (n_a.vd[17][index] + n_a.vd[18][index])
           + 2.f*((n_a.vd[5][index] + n_a.vd[6][index]) + (n_a.vd[7][index] + n_a.vd[8][index])
           + (n_a.vd[9][index] + n_a.vd[10][index]));

}
/*-------------------------------------------------------*/
/**lb_relax_modes, means collision*/
/*@{
 * @param n_a		Pointer to local node residing in array a (Input)
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Input/Output)
 * @param node_f	Pointer to local node force (Input)
}*/
/*-------------------------------------------------------*/
__device__ void relax_modes(float *mode, unsigned int index, LB_node_force_gpu node_f){

  float Rho = mode[0] + para.rho*para.agrid*para.agrid*para.agrid;
  float j[3], pi_eq[6];

  /* re-construct the real density
  * remember that the populations are stored as differences to their
  * equilibrium value */

  j[0] = mode[1];
  j[1] = mode[2];
  j[2] = mode[3];

  /* if forces are present, the momentum density is redefined to
  * inlcude one half-step of the force action.  See the
  * Chapman-Enskog expansion in [Ladd & Verberg]. */

  j[0] += 0.5f*node_f.force[0][index];
  j[1] += 0.5f*node_f.force[1][index];
  j[2] += 0.5f*node_f.force[2][index];

  /* equilibrium part of the stress modes (eq13 schiller)*/
  pi_eq[0] = ((j[0]*j[0])+(j[1]*j[1])+(j[2]*j[2]))/Rho;
  pi_eq[1] = ((j[0]*j[0])-(j[1]*j[1]))/Rho;
  pi_eq[2] = (((j[0]*j[0])+(j[1]*j[1])+(j[2]*j[2])) - 3.0f*(j[2]*j[2]))/Rho;
  pi_eq[3] = j[0]*j[1]/Rho;
  pi_eq[4] = j[0]*j[2]/Rho;
  pi_eq[5] = j[1]*j[2]/Rho;

  /* relax the stress modes (eq14 schiller)*/
  mode[4] = pi_eq[0] + para.gamma_bulk*(mode[4] - pi_eq[0]);
  mode[5] = pi_eq[1] + para.gamma_shear*(mode[5] - pi_eq[1]);
  mode[6] = pi_eq[2] + para.gamma_shear*(mode[6] - pi_eq[2]);
  mode[7] = pi_eq[3] + para.gamma_shear*(mode[7] - pi_eq[3]);
  mode[8] = pi_eq[4] + para.gamma_shear*(mode[8] - pi_eq[4]);
  mode[9] = pi_eq[5] + para.gamma_shear*(mode[9] - pi_eq[5]);

  /* relax the ghost modes (project them out) */
  /* ghost modes have no equilibrium part due to orthogonality */
  mode[10] = para.gamma_odd*mode[10];
  mode[11] = para.gamma_odd*mode[11];
  mode[12] = para.gamma_odd*mode[12];
  mode[13] = para.gamma_odd*mode[13];
  mode[14] = para.gamma_odd*mode[14];
  mode[15] = para.gamma_odd*mode[15];
  mode[16] = para.gamma_even*mode[16];
  mode[17] = para.gamma_even*mode[17];
  mode[18] = para.gamma_even*mode[18];

}
/*-------------------------------------------------------*/
/**thermalization of the modes with gaussian random numbers*/
/*@{
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Input/Output)
 * @param *rn		Pointer to randomnumber array of the local node
 */
/*-------------------------------------------------------*/
__device__ void thermalize_modes(float *mode, unsigned int index, LB_randomnr_gpu *rn){

  float rootrho = sqrt(mode[0]+para.rho*para.agrid*para.agrid*para.agrid);

  /* stress modes */
  gaussian_random(rn);
  mode[4] += rootrho*(para.mu*(2.f/3.f)*(1.f-(para.gamma_bulk*para.gamma_bulk))) * rn->randomnr[1];
  mode[5] += rootrho*(para.mu*(4.f/9.f)*(1.f-(para.gamma_shear*para.gamma_shear))) * rn->randomnr[0];

  gaussian_random(rn);
  mode[6] += rootrho*(para.mu*(4.f/3.f)*(1.f-(para.gamma_shear*para.gamma_shear))) * rn->randomnr[1];
  mode[7] += rootrho*(para.mu*(1.f/9.f)*(1.f-(para.gamma_shear*para.gamma_shear))) * rn->randomnr[0];

  gaussian_random(rn);
  mode[8] += rootrho*(para.mu*(1.f/9.f)*(1.f-(para.gamma_shear*para.gamma_shear))) * rn->randomnr[1];
  mode[9] += rootrho*(para.mu*(1.f/9.f)*(1.f-(para.gamma_shear*para.gamma_shear))) * rn->randomnr[0];
 
  /* ghost modes */
  gaussian_random(rn);
  mode[10] += rootrho*(para.mu*(2.f/3.f)) * rn->randomnr[1];
  mode[11] += rootrho*(para.mu*(2.f/3.f)) * rn->randomnr[0];

  gaussian_random(rn);
  mode[12] += rootrho*(para.mu*(2.f/3.f)) * rn->randomnr[1];
  mode[13] += rootrho*(para.mu*(2.f/9.f)) * rn->randomnr[0];

  gaussian_random(rn);
  mode[14] += rootrho*(para.mu*(2.f/9.f)) * rn->randomnr[1];
  mode[15] += rootrho*(para.mu*(2.f/9.f)) * rn->randomnr[0];

  gaussian_random(rn);
  mode[16] += rootrho*(para.mu*(2.f)) * rn->randomnr[1];
  mode[17] += rootrho*(para.mu*(4.f/9.f)) * rn->randomnr[0];

  gaussian_random(rn);
  mode[18] += rootrho*(para.mu*(4.f/3.f)) * rn->randomnr[1];

}
/*-------------------------------------------------------*/
/**normalization of the modes need befor backtransformation into velocity space*/
/*@{
 * @param mode		Pointer to the local register values mode (Input/Output)
}*/
/*-------------------------------------------------------*/
__device__ void normalize_modes(float* mode){

  /* normalization factors enter in the back transformation */
  /* the following values are the (weighted) lengths of the vectors */
  mode[0] *= 1.f;
  mode[1] *= 3.f;
  mode[2] *= 3.f;
  mode[3] *= 3.f;
  mode[4] *= 3.f/2.f;
  mode[5] *= 9.f/4.f;
  mode[6] *= 3.f/4.f;
  mode[7] *= 9.f;
  mode[8] *= 9.f;
  mode[9] *= 9.f;
  mode[10] *= 3.f/2.f;
  mode[11] *= 3.f/2.f;
  mode[12] *= 3.f/2.f;
  mode[13] *= 9.f/2.f;
  mode[14] *= 9.f/2.f;
  mode[15] *= 9.f/2.f;
  mode[16] *= 1.f/2.f;
  mode[17] *= 9.f/4.f;
  mode[18] *= 3.f/4.f;

}
/*-------------------------------------------------------*/
/**backtransformation from modespace to desityspace and streaming with the push method using pbc*/
/*@{
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Input)
 * @param *n_b		Pointer to local node residing in array b (Output)
}*/
/*-------------------------------------------------------*/
__device__ void calc_n_from_modes_push(LB_nodes_gpu n_b, float *mode, unsigned int index){

  unsigned int xyz[3];
  index_to_xyz(index, xyz);
  unsigned int x = xyz[0];
  unsigned int y = xyz[1];
  unsigned int z = xyz[2];

  n_b.vd[0][x + para.dim_x*y + para.dim_x*para.dim_y*z] = 1.f/3.f * (mode[0] - mode[4] + mode[16]);
  n_b.vd[1][(x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = 1.f/18.f * (mode[0] + mode[1] + mode[5] + mode[6] - mode[17] - mode[18] - 2.f*(mode[10] + mode[16]));
  n_b.vd[2][(para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = 1.f/18.f * (mode[0] - mode[1] + mode[5] + mode[6] - mode[17] - mode[18] + 2.f*(mode[10] - mode[16]));
  n_b.vd[3][x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/18.f * (mode[0] + mode[2] - mode[5] + mode[6] + mode[17] - mode[18] - 2.f*(mode[11] + mode[16]));
  n_b.vd[4][x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/18.f * (mode[0] - mode[2] - mode[5] + mode[6] + mode[17] - mode[18] + 2.f*(mode[11] - mode[16]));
  n_b.vd[5][x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/18.f * (mode[0] + mode[3] - 2.f*(mode[6] + mode[12] + mode[16] - mode[18]));
  n_b.vd[6][x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/18.f * (mode[0] - mode[3] - 2.f*(mode[6] - mode[12] + mode[16] - mode[18]));
  n_b.vd[7][(x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/36.f * (mode[0] + mode[1] + mode[2] + mode[4] + 2.f*mode[6] + mode[7] + mode[10] + mode[11] + mode[13] + mode[14] + mode[16] + 2.f*mode[18]);
  n_b.vd[8][(para.dim_x+x-1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/36.f * (mode[0] - mode[1] - mode[2] + mode[4] + 2.f*mode[6] + mode[7] - mode[10] - mode[11] - mode[13] - mode[14] + mode[16] + 2.f*mode[18]);
  n_b.vd[9][(x+1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/36.f * (mode[0] + mode[1] - mode[2] + mode[4] + 2.f*mode[6] - mode[7] + mode[10] - mode[11] + mode[13] - mode[14] + mode[16] + 2.f*mode[18]);
  n_b.vd[10][(para.dim_x+x-1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/36.f * (mode[0] - mode[1] + mode[2] + mode[4] + 2.f*mode[6] - mode[7] - mode[10] + mode[11] - mode[13] + mode[14] + mode[16] + 2.f*mode[18]);
  n_b.vd[11][(x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/36.f * (mode[0] + mode[1] + mode[3] + mode[4] + mode[5] - mode[6] + mode[8] + mode[10] + mode[12] - mode[13] + mode[15] + mode[16] + mode[17] - mode[18]);
  n_b.vd[12][(para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/36.f * (mode[0] - mode[1] - mode[3] + mode[4] + mode[5] - mode[6] + mode[8] - mode[10] - mode[12] + mode[13] - mode[15] + mode[16] + mode[17] - mode[18]);
  n_b.vd[13][(x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/36.f * (mode[0] + mode[1] - mode[3] + mode[4] + mode[5] - mode[6] - mode[8] + mode[10] - mode[12] - mode[13] - mode[15] + mode[16] + mode[17] - mode[18]);
  n_b.vd[14][(para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/36.f * (mode[0] - mode[1] + mode[3] + mode[4] + mode[5] - mode[6] - mode[8] - mode[10] + mode[12] + mode[13] + mode[15] + mode[16] + mode[17] - mode[18]);
  n_b.vd[15][x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/36.f * (mode[0] + mode[2] + mode[3] + mode[4] - mode[5] - mode[6] + mode[9] + mode[11] + mode[12] - mode[14] - mode[15] + mode[16] - mode[17] - mode[18]);
  n_b.vd[16][x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/36.f * (mode[0] - mode[2] - mode[3] + mode[4] - mode[5] - mode[6] + mode[9] - mode[11] - mode[12] + mode[14] + mode[15] + mode[16] - mode[17] - mode[18]);
  n_b.vd[17][x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/36.f * (mode[0] + mode[2] - mode[3] + mode[4] - mode[5] - mode[6] - mode[9] + mode[11] - mode[12] - mode[14] + mode[15] + mode[16] - mode[17] - mode[18]);
  n_b.vd[18][x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/36.f * (mode[0] - mode[2] + mode[3] + mode[4] - mode[5] - mode[6] - mode[9] - mode[11] + mode[12] + mode[14] - mode[15] + mode[16] - mode[17] - mode[18]);

}
/*-------------------------------------------------------*/
/** Bounce back boundary conditions.
 * The populations that have propagated into a boundary node
 * are bounced back to the node they came from. This results
 * in no slip boundary conditions.
 *
 * [cf. Ladd and Verberg, J. Stat. Phys. 104(5/6):1191-1251, 2001]
 */
/*@{
 * @param index			node index / thread index (Input)
 * @param n_b			Pointer to local node residing in array b (Input)
 * @param n_a			Pointer to local node residing in array a (Output) (temp stored in buffer a)
}*/
/*-------------------------------------------------------*/
__device__ void bounce_back_read(LB_nodes_gpu n_b, LB_nodes_gpu n_a, unsigned int index){
    
  unsigned int xyz[3];

  if(n_b.boundary[index] == 1){
    index_to_xyz(index, xyz);
    unsigned int x = xyz[0];
    unsigned int y = xyz[1];
    unsigned int z = xyz[2];

    /* stream vd from boundary node back to origin node */
    n_a.vd[1][(x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = n_b.vd[2][index];
    n_a.vd[2][(para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = n_b.vd[1][index];
    n_a.vd[3][x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_b.vd[4][index];
    n_a.vd[4][x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_b.vd[3][index];
    n_a.vd[5][x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_b.vd[6][index];
    n_a.vd[6][x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_b.vd[5][index];
    n_a.vd[7][(x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_b.vd[8][index];
    n_a.vd[8][(para.dim_x+x-1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_b.vd[7][index];
    n_a.vd[9][(x+1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_b.vd[10][index];
    n_a.vd[10][(para.dim_x+x-1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_b.vd[9][index];
    n_a.vd[11][(x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_b.vd[12][index];
    n_a.vd[12][(para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_b.vd[11][index]; 
    n_a.vd[13][(x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_b.vd[14][index]; 
    n_a.vd[14][(para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_b.vd[13][index]; 
    n_a.vd[15][x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_b.vd[16][index];
    n_a.vd[16][x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_b.vd[15][index];
    n_a.vd[17][x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_b.vd[18][index]; 
    n_a.vd[18][x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_b.vd[17][index];
  }
}
/*-------------------------------------------------------*/
/*@{
 * @param index			node index / thread index (Input)
 * @param n_b			Pointer to local node residing in array b (Input)
 * @param n_a			Pointer to local node residing in array a (Output) (temp stored in buffer a)
}*/
/*-------------------------------------------------------*/
__device__ void bounce_back_write(LB_nodes_gpu n_b, LB_nodes_gpu n_a, unsigned int index){

  unsigned int xyz[3];

  if(n_b.boundary[index] == 1){
    index_to_xyz(index, xyz);
    unsigned int x = xyz[0];
    unsigned int y = xyz[1];
    unsigned int z = xyz[2];

    n_b.vd[1][(x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = n_a.vd[1][(x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z];
    n_b.vd[2][(para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = n_a.vd[2][(para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z];
    n_b.vd[3][x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[3][x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[4][x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[4][x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[5][x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[5][x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
    n_b.vd[6][x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[6][x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[7][(x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[7][(x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[8][(para.dim_x+x-1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[8][(para.dim_x+x-1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[9][(x+1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[9][(x+1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[10][(para.dim_x+x-1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[10][(para.dim_x+x-1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[11][(x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[11][(x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
    n_b.vd[12][(para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[12][(para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[13][(x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[13][(x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[14][(para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[14][(para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
    n_b.vd[15][x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[15][x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
    n_b.vd[16][x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[16][x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[17][x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[17][x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[18][x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[18][x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
  }
}

/*-------------------------------------------------------*/
/**used for reset the field to create qickanddirty non periodic bc !!only for advanced users!!*/
/*@{
 * @param index			node index / thread index (Input)
 * @param n_a			Pointer to local node residing in array a (Input)
 * @param *n_b			Pointer to local node residing in array b (Output)
}*/
/*-------------------------------------------------------*/
__device__ void reset_pop(LB_nodes_gpu n_b, LB_nodes_gpu n_a, unsigned int index){	

  //float avg_rho = para.rho*para.agrid*para.agrid*para.agrid;
  /* delete populations */
#if 1
  if(n_b.boundary[index] == 2){
    n_b.vd[0][index] = n_b.vd[0][index+1];
    n_b.vd[1][index] = n_b.vd[1][index+1];
    n_b.vd[2][index] = n_b.vd[2][index+1];
    n_b.vd[3][index] = n_b.vd[3][index+1];
    n_b.vd[4][index] = n_b.vd[4][index+1];
    n_b.vd[5][index] = n_b.vd[5][index+1];
    n_b.vd[6][index] = n_b.vd[6][index+1];
    n_b.vd[7][index]  = n_b.vd[7][index+1];
    n_b.vd[8][index]  = n_b.vd[8][index+1];
    n_b.vd[9][index]  = n_b.vd[9][index+1];
    n_b.vd[10][index] = n_b.vd[10][index+1];
    n_b.vd[11][index] = n_b.vd[11][index+1];
    n_b.vd[12][index] = n_b.vd[12][index+1];
    n_b.vd[13][index] = n_b.vd[13][index+1];
    n_b.vd[14][index] = n_b.vd[14][index+1];
    n_b.vd[15][index] = n_b.vd[15][index+1];
    n_b.vd[16][index] = n_b.vd[16][index+1];
    n_b.vd[17][index] = n_b.vd[17][index+1];
    n_b.vd[18][index] = n_b.vd[18][index+1];
  }
#endif
}
/*-------------------------------------------------------*/
/** add of (external) forces within the modespace, needed for particle-interaction */
/*@{
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Input/Output)
 * @param node_f	Pointer to local node force (Input)
}*/
/*-------------------------------------------------------*/
__device__ void apply_forces(unsigned int index, float *mode, LB_node_force_gpu node_f) {

  float Rho, u[3], C[6];
  Rho = mode[0] + para.rho*para.agrid*para.agrid*para.agrid;

  /* hydrodynamic momentum density is redefined when forces present */
  u[0] = (mode[1] + 0.5f*node_f.force[0][index])/Rho;
  u[1] = (mode[2] + 0.5f*node_f.force[1][index])/Rho;
  u[2] = (mode[3] + 0.5f*node_f.force[2][index])/Rho;

  C[0] = (1.f + para.gamma_bulk)*u[0]*node_f.force[0][index] + 1.f/3.f*(para.gamma_bulk-para.gamma_shear)*(u[0]*node_f.force[0][index] + u[1]*node_f.force[1][index] + u[2]*node_f.force[2][index]);
  C[2] = (1.f + para.gamma_bulk)*u[1]*node_f.force[1][index] + 1.f/3.f*(para.gamma_bulk-para.gamma_shear)*(u[0]*node_f.force[0][index] + u[1]*node_f.force[1][index] + u[2]*node_f.force[2][index]);
  C[5] = (1.f + para.gamma_bulk)*u[2]*node_f.force[2][index] + 1.f/3.f*(para.gamma_bulk-para.gamma_shear)*(u[0]*node_f.force[0][index] + u[1]*node_f.force[1][index] + u[2]*node_f.force[2][index]);
  C[1] = 1.f/2.f*(1.f+para.gamma_shear)*(u[0]*node_f.force[1][index]+u[1]*node_f.force[0][index]);
  C[3] = 1.f/2.f*(1.f+para.gamma_shear)*(u[0]*node_f.force[2][index]+u[2]*node_f.force[0][index]);
  C[4] = 1.f/2.f*(1.f+para.gamma_shear)*(u[1]*node_f.force[2][index]+u[2]*node_f.force[1][index]);

  /* update momentum modes */
  mode[1] += node_f.force[0][index];
  mode[2] += node_f.force[1][index];
  mode[3] += node_f.force[2][index];
  	
  /* update stress modes */
  mode[4] += C[0] + C[2] + C[5];
  mode[5] += C[0] - C[2];
  mode[6] += C[0] + C[2] - 2.f*C[5];
  mode[7] += C[1];
  mode[8] += C[3];
  mode[9] += C[4];

#ifdef EXTERNAL_FORCES
  if(para.external_force){
    node_f.force[0][index] = para.ext_force[0]*para.agrid*para.agrid*para.tau*para.tau;
    node_f.force[1][index] = para.ext_force[1]*para.agrid*para.agrid*para.tau*para.tau;
    node_f.force[2][index] = para.ext_force[2]*para.agrid*para.agrid*para.tau*para.tau;
  }
#else
  /* reset force */
  node_f.force[0][index] = 0.f;
  node_f.force[1][index] = 0.f;
  node_f.force[2][index] = 0.f;
#endif
}
/*-------------------------------------------------------*/
/**function used to calc physical values of every node*/
/*@{
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Input)
 * @param n_a		Pointer to local node residing in array a for boundary flag(Input)
 * @param *d_v		Pointer to local device values (Input/Output)
}*/
/*-------------------------------------------------------*/
__device__ void calc_values(LB_nodes_gpu n_a, float *mode, LB_values_gpu *d_v, unsigned int index, unsigned int singlenode){

  float Rho = mode[0] + para.rho*para.agrid*para.agrid*para.agrid;
	
  /**implemented due to the problem of division via zero*/
  if(n_a.boundary[index] == 1){
    Rho = 1.0f;
    mode[1] = 0.f;
    mode[2] = 0.f;
    mode[3] = 0.f;
  }

  if(singlenode == 1){
    d_v[0].rho = Rho;
    d_v[0].v[0] = mode[1]/Rho;
    d_v[0].v[1] = mode[2]/Rho;
    d_v[0].v[2] = mode[3]/Rho;
  }
  else{
    d_v[index].rho = Rho;
    d_v[index].v[0] = mode[1]/Rho;
    d_v[index].v[1] = mode[2]/Rho;
    d_v[index].v[2] = mode[3]/Rho;
  }
#if 0
  if(singlenode == 1){
    /* equilibrium part of the stress modes */
    /**to print out the stress tensor entries, ensure that in lbgpu.h struct the values are available*/
    d_v[0].pi[0] = ((mode[1]*mode[1]) + (mode[2]*mode[2]) + (mode[3]*mode[3]))/para.rho;
    d_v[0].pi[1] = ((mode[1]*mode[1]) - (mode[2]*mode[2]))/para.rho;
    d_v[0].pi[2] = ((mode[1]*mode[1]) + (mode[2]*mode[2])  + (mode[3]*mode[3])) - 3.0f*(mode[3]*mode[3]))/para.rho;
    d_v[0].pi[3] = mode[1]*mode[2]/para.rho;
    d_v[0].pi[4] = mode[1]*mode[3]/para.rho;
    d_v[0].pi[5] = mode[2]*mode[3]/para.rho;
  else{
    d_v[index].pi[0] = ((mode[1]*mode[1]) + (mode[2]*mode[2]) + (mode[3]*mode[3]))/para.rho;
    d_v[index].pi[1] = ((mode[1]*mode[1]) - (mode[2]*mode[2]))/para.rho;
    d_v[index].pi[2] = ((mode[1]*mode[1]) + (mode[2]*mode[2])  + (mode[3]*mode[3])) - 3.0f*(mode[3]*mode[3]))/para.rho;
    d_v[index].pi[3] = mode[1]*mode[2]/para.rho;
    d_v[index].pi[4] = mode[1]*mode[3]/para.rho;
    d_v[index].pi[5] = mode[2]*mode[3]/para.rho;
  }
#endif
}
/*-------------------------------------------------------*/
/*@{
 * @param node_index	node index around (8) particle (Input)
 * @param *mode			Pointer to the local register values mode (Output)
 * @param n_a			Pointer to local node residing in array a(Input)
}*/
/*-------------------------------------------------------*/
__device__ void calc_mode(float *mode, LB_nodes_gpu n_a, unsigned int node_index){
	
  /* mass mode */
  mode[0] = n_a.vd[0][node_index] + n_a.vd[1][node_index] + n_a.vd[2][node_index]
          + n_a.vd[3][node_index] + n_a.vd[4][node_index] + n_a.vd[5][node_index]
          + n_a.vd[6][node_index] + n_a.vd[7][node_index] + n_a.vd[8][node_index]
          + n_a.vd[9][node_index] + n_a.vd[10][node_index] + n_a.vd[11][node_index] + n_a.vd[12][node_index]
          + n_a.vd[13][node_index] + n_a.vd[14][node_index] + n_a.vd[15][node_index] + n_a.vd[16][node_index]
          + n_a.vd[17][node_index] + n_a.vd[18][node_index];

  /* momentum modes */
  mode[1] = (n_a.vd[1][node_index] - n_a.vd[2][node_index]) + (n_a.vd[7][node_index] - n_a.vd[8][node_index])
          + (n_a.vd[9][node_index] - n_a.vd[10][node_index]) + (n_a.vd[11][node_index] - n_a.vd[12][node_index])
          + (n_a.vd[13][node_index] - n_a.vd[14][node_index]);
  mode[2] = (n_a.vd[3][node_index] - n_a.vd[4][node_index]) + (n_a.vd[7][node_index] - n_a.vd[8][node_index])
          - (n_a.vd[9][node_index] - n_a.vd[10][node_index]) + (n_a.vd[15][node_index] - n_a.vd[16][node_index])
          + (n_a.vd[17][node_index] - n_a.vd[18][node_index]);
  mode[3] = (n_a.vd[5][node_index] - n_a.vd[6][node_index]) + (n_a.vd[11][node_index] - n_a.vd[12][node_index])
          - (n_a.vd[13][node_index] - n_a.vd[14][node_index]) + (n_a.vd[15][node_index] - n_a.vd[16][node_index])
          - (n_a.vd[17][node_index] - n_a.vd[18][node_index]);
}
/*********************************************************/
/** \name Coupling part */
/*********************************************************/
/*-------------------------------------------------------*/
/**(Eq. (12) Ahlrichs and Duenweg, JCP 111(17):8225 (1999))*/
/*@{
 * @param n_a				Pointer to local node residing in array a (Input)
 * @param *delta			Pointer for the weighting of particle position (Output)
 * @param *delta_j			Pointer for the weighting of particle momentum (Output)
 * @param *particle_data	Pointer to the particle position and velocity (Input)
 * @param *particle_force	Pointer to the particle force (Input)
 * @param part_index		particle id / thread id (Input)
 * @param *rn				Pointer to randomnumber array of the particle
 * @param node_index		node index around (8) particle (Output)
}*/
/*-------------------------------------------------------*/
__device__ void calc_viscous_force(LB_nodes_gpu n_a, float *delta, LB_particle_gpu *particle_data, LB_particle_force_gpu *particle_force, unsigned int part_index, LB_randomnr_gpu *rn_part, float *delta_j, unsigned int *node_index){
	
  float mode[4];
  unsigned int my_left[3];
  float interpolated_u1, interpolated_u2, interpolated_u3;
  float Rho;
  interpolated_u1 = interpolated_u2 = interpolated_u3 = 0.f;

  float temp_delta[6];
  float temp_delta_half[6];

/** see ahlrichs + duennweg page 8227 equ (10) and (11) */
  #pragma unroll
  for(int i=0; i<3; ++i){
    float scaledpos = particle_data[part_index].p[i]/para.agrid;
    my_left[i] = (unsigned int)(floorf(scaledpos));
    temp_delta[3+i] = scaledpos - my_left[i];
    temp_delta[i] = 1.f - temp_delta[3+i];
    /**further value used for interpolation of fluid velocity at part pos near boundaries */
    temp_delta_half[3+i] = (scaledpos - my_left[i])*2.f;
    temp_delta_half[i] = 2.f - temp_delta_half[3+i];
  }

  delta[0] = temp_delta[0] * temp_delta[1] * temp_delta[2];
  delta[1] = temp_delta[3] * temp_delta[1] * temp_delta[2];
  delta[2] = temp_delta[0] * temp_delta[4] * temp_delta[2];
  delta[3] = temp_delta[3] * temp_delta[4] * temp_delta[2];
  delta[4] = temp_delta[0] * temp_delta[1] * temp_delta[5];
  delta[5] = temp_delta[3] * temp_delta[1] * temp_delta[5];
  delta[6] = temp_delta[0] * temp_delta[4] * temp_delta[5];
  delta[7] = temp_delta[3] * temp_delta[4] * temp_delta[5];

  unsigned int x = my_left[0];
  unsigned int y = my_left[1];
  unsigned int z = my_left[2];

  node_index[0] = x                + para.dim_x*y                  + para.dim_x*para.dim_y*z;
  node_index[1] = (x+1)%para.dim_x + para.dim_x*y                  + para.dim_x*para.dim_y*z;
  node_index[2] = x                + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z;
  node_index[3] = (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z;
  node_index[4] = x                + para.dim_x*y                  + para.dim_x*para.dim_y*((z+1)%para.dim_z);
  node_index[5] = (x+1)%para.dim_x + para.dim_x*y                  + para.dim_x*para.dim_y*((z+1)%para.dim_z);
  node_index[6] = x                + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z);
  node_index[7] = (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z);
	
#if 0
	/** calc of the interpolated verlocity at the position of the particle !!!still under investigation and development!!!*/
  if(n_a.boundary[node_index[0]] == 1){
    delta[1] = temp_delta_half[3] * temp_delta[1] * temp_delta[2];
    delta[2] = temp_delta[0] * temp_delta_half[4] * temp_delta[2];
    delta[4] = temp_delta[0] * temp_delta[1] * temp_delta_half[5];
  }
  if(n_a.boundary[node_index[1]] == 1){		
    delta[0] = temp_delta_half[0] * temp_delta[1] * temp_delta[2];
    delta[3] = temp_delta[3] * temp_delta_half[4] * temp_delta[2];
    delta[5] = temp_delta[3] * temp_delta[1] * temp_delta_half[5];
  }
  if(n_a.boundary[node_index[2]] == 1){		
    delta[0] = temp_delta_half[0] * temp_delta[1] * temp_delta[2];
    delta[3] = temp_delta[3] * temp_delta_half[4] * temp_delta[2];
    delta[6] = temp_delta[0] * temp_delta[4] * temp_delta_half[5];
  }
  if(n_a.boundary[node_index[3]] == 1){		
    delta[1] = temp_delta[3] * temp_delta_half[1] * temp_delta[2];
    delta[2] = temp_delta_half[0] * temp_delta[4] * temp_delta[2];
    delta[7] = temp_delta[3] * temp_delta[4] * temp_delta_half[5];
  }
  if(n_a.boundary[node_index[4]] == 1){		
    delta[0] = temp_delta[0] * temp_delta[1] * temp_delta_half[2];
    delta[5] = temp_delta_half[3] * temp_delta[1] * temp_delta[5];
    delta[6] = temp_delta[0] * temp_delta_half[4] * temp_delta[5];
  }
  if(n_a.boundary[node_index[5]] == 1){		
    delta[1] = temp_delta[3] * temp_delta[1] * temp_delta_half[2];
    delta[4] = temp_delta_half[0] * temp_delta[1] * temp_delta[5];
    delta[7] = temp_delta[3] * temp_delta_half[4] * temp_delta[5];
  }
  if(n_a.boundary[node_index[6]] == 1){		
    delta[2] = temp_delta[0] * temp_delta[4] * temp_delta_half[2];
    delta[4] = temp_delta[0] * temp_delta_half[1] * temp_delta[5];
    delta[7] = temp_delta_half[3] * temp_delta[4] * temp_delta[5];
  }
  if(n_a.boundary[node_index[7]] == 1){		
    delta[3] = temp_delta[3] * temp_delta[4] * temp_delta_half[2];
    delta[5] = temp_delta[3] * temp_delta_half[1] * temp_delta[5];
    delta[6] = temp_delta_half[0] * temp_delta[4] * temp_delta[5];
  }

  if(n_a.boundary[node_index[0]] == 1)delta[0] = 0.f;

  if(n_a.boundary[node_index[1]] == 1)delta[1] = 0.f;

  if(n_a.boundary[node_index[2]] == 1)delta[2] = 0.f;

  if(n_a.boundary[node_index[3]] == 1)delta[3] = 0.f;

  if(n_a.boundary[node_index[4]] == 1)delta[4] = 0.f;

  if(n_a.boundary[node_index[5]] == 1)delta[5] = 0.f;

  if(n_a.boundary[node_index[6]] == 1)delta[6] = 0.f;

  if(n_a.boundary[node_index[7]] == 1)delta[7] = 0.f;
#endif

 #pragma unroll
  for(int i=0; i<8; ++i){
    calc_mode(mode, n_a, node_index[i]);
    Rho = mode[0] + para.rho*para.agrid*para.agrid*para.agrid;	
    interpolated_u1 += delta[i]*mode[1]/(Rho);
    interpolated_u2 += delta[i]*mode[2]/(Rho);
    interpolated_u3 += delta[i]*mode[3]/(Rho);
  }

	/* calculate viscous force
	* take care to rescale velocities with time_step and transform to MD units
 	* (Eq. (9) Ahlrichs and Duenweg, JCP 111(17):8225 (1999)) */
#ifdef LB_ELECTROHYDRODYNAMICS
  particle_force[part_index].f[0] = - para.friction * (particle_data[part_index].v[0]/para.time_step - interpolated_u1*para.agrid/para.tau - particle_data[part_index].mu_E[0]);
  particle_force[part_index].f[1] = - para.friction * (particle_data[part_index].v[1]/para.time_step - interpolated_u2*para.agrid/para.tau - particle_data[part_index].mu_E[1]);
  particle_force[part_index].f[2] = - para.friction * (particle_data[part_index].v[2]/para.time_step - interpolated_u3*para.agrid/para.tau - particle_data[part_index].mu_E[2]);
#else
  particle_force[part_index].f[0] = - para.friction * (particle_data[part_index].v[0]/para.time_step - interpolated_u1*para.agrid/para.tau);
  particle_force[part_index].f[1] = - para.friction * (particle_data[part_index].v[1]/para.time_step - interpolated_u2*para.agrid/para.tau);
  particle_force[part_index].f[2] = - para.friction * (particle_data[part_index].v[2]/para.time_step - interpolated_u3*para.agrid/para.tau);
#endif
	/** add stochastik force of zero mean (Ahlrichs, Duennweg equ. 15)*/
#ifdef GAUSSRANDOM
  gaussian_random(rn_part);
  particle_force[part_index].f[0] += para.lb_coupl_pref2*rn_part->randomnr[0];
  particle_force[part_index].f[1] += para.lb_coupl_pref2*rn_part->randomnr[1];
  gaussian_random(rn_part);
  particle_force[part_index].f[2] += para.lb_coupl_pref2*rn_part->randomnr[0];
#else
  random_01(rn_part);
  particle_force[part_index].f[0] += para.lb_coupl_pref*(rn_part->randomnr[0]-0.5f);
  particle_force[part_index].f[1] += para.lb_coupl_pref*(rn_part->randomnr[1]-0.5f);
  random_01(rn_part);
  particle_force[part_index].f[2] += para.lb_coupl_pref*(rn_part->randomnr[0]-0.5f);
#endif
	  
  /* delta_j for transform momentum transfer to lattice units which is done in calc_node_force
  (Eq. (12) Ahlrichs and Duenweg, JCP 111(17):8225 (1999)) */
  delta_j[0] = - particle_force[part_index].f[0]*para.time_step*para.tau/para.agrid;
  delta_j[1] = - particle_force[part_index].f[1]*para.time_step*para.tau/para.agrid;
  delta_j[2] = - particle_force[part_index].f[2]*para.time_step*para.tau/para.agrid;  	
															  																	  
}
/*-------------------------------------------------------*/
/**calcutlation of the node force caused by the particles, with atomicadd due to avoiding race conditions 
	(Eq. (14) Ahlrichs and Duenweg, JCP 111(17):8225 (1999))*/
/*@{

 * @param *delta			Pointer for the weighting of particle position (Input)
 * @param *delta_j			Pointer for the weighting of particle momentum (Input)
 * @param node_index		node index around (8) particle (Input)
 * @param node_f    		Pointer to the node force (Output).
}*/
/*-------------------------------------------------------*/
__device__ void calc_node_force(float *delta, float *delta_j, unsigned int *node_index, LB_node_force_gpu node_f){

  atomicadd(&(node_f.force[0][node_index[0]]), (delta[0]*delta_j[0]));
  atomicadd(&(node_f.force[1][node_index[0]]), (delta[0]*delta_j[1]));
  atomicadd(&(node_f.force[2][node_index[0]]), (delta[0]*delta_j[2]));

  atomicadd(&(node_f.force[0][node_index[1]]), (delta[1]*delta_j[0]));
  atomicadd(&(node_f.force[1][node_index[1]]), (delta[1]*delta_j[1]));
  atomicadd(&(node_f.force[2][node_index[1]]), (delta[1]*delta_j[2]));

  atomicadd(&(node_f.force[0][node_index[2]]), (delta[2]*delta_j[0]));
  atomicadd(&(node_f.force[1][node_index[2]]), (delta[2]*delta_j[1]));
  atomicadd(&(node_f.force[2][node_index[2]]), (delta[2]*delta_j[2]));

  atomicadd(&(node_f.force[0][node_index[3]]), (delta[3]*delta_j[0]));
  atomicadd(&(node_f.force[1][node_index[3]]), (delta[3]*delta_j[1]));
  atomicadd(&(node_f.force[2][node_index[3]]), (delta[3]*delta_j[2]));

  atomicadd(&(node_f.force[0][node_index[4]]), (delta[4]*delta_j[0]));
  atomicadd(&(node_f.force[1][node_index[4]]), (delta[4]*delta_j[1]));
  atomicadd(&(node_f.force[2][node_index[4]]), (delta[4]*delta_j[2]));

  atomicadd(&(node_f.force[0][node_index[5]]), (delta[5]*delta_j[0]));
  atomicadd(&(node_f.force[1][node_index[5]]), (delta[5]*delta_j[1]));
  atomicadd(&(node_f.force[2][node_index[5]]), (delta[5]*delta_j[2]));

  atomicadd(&(node_f.force[0][node_index[6]]), (delta[6]*delta_j[0]));
  atomicadd(&(node_f.force[1][node_index[6]]), (delta[6]*delta_j[1]));
  atomicadd(&(node_f.force[2][node_index[6]]), (delta[6]*delta_j[2]));

  atomicadd(&(node_f.force[0][node_index[7]]), (delta[7]*delta_j[0]));
  atomicadd(&(node_f.force[1][node_index[7]]), (delta[7]*delta_j[1]));
  atomicadd(&(node_f.force[2][node_index[7]]), (delta[7]*delta_j[2]));

}
/*-------------------------------------------------------*/
/**additional check if the particles are within the box */
/*@{
 * @param *particle_data	Pointer to the particle data (Input).
 * @param part_index		index of the particle == thread index (Input).		
}*/
/*-------------------------------------------------------*/
__device__ void check_part_posis(LB_particle_gpu *particle_data, unsigned int part_index){

  if(particle_data[part_index].p[0]/para.agrid < 0.f || particle_data[part_index].p[0]/para.agrid > para.dim_x){
    printf("particle out of box! (dim_x) \t %u \t %f \n", part_index, particle_data[part_index].p[0]); 
  }
  if(particle_data[part_index].p[1]/para.agrid < 0.f || particle_data[part_index].p[1]/para.agrid > para.dim_y){
    printf("particle out of box! (dim_y) \t %u \t %f \n", part_index, particle_data[part_index].p[1]); 
  }
  if(particle_data[part_index].p[2]/para.agrid < 0.f || particle_data[part_index].p[2]/para.agrid > para.dim_z){
    printf("particle out of box! (dim_z) \t %u \t %f \n", part_index, particle_data[part_index].p[2]); 
  }
}
/*-------------------------------------------------------*/
/**kernel to calculate local populations from hydrodynamic fields given by the tcl values.
 *
 * The mapping is given in terms of the equilibrium distribution.
 *
 * Eq. (2.15) Ladd, J. Fluid Mech. 271, 295-309 (1994)
 * Eq. (4) in Berk Usta, Ladd and Butler, JCP 122, 094902 (2005)
 *
 * @param n_a		 Pointer to the lattice site (Input).
 * @param node_f    Pointer to the node force (Input).
 * @param *gpu_check additional check if gpu kernel are executed(Input).
 */
/*-------------------------------------------------------*/
__global__ void calc_n_equilibrium(LB_nodes_gpu n_a, LB_node_force_gpu node_f, int *gpu_check) {

  unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){

    /*temp gesetzt aus lb_reinit_fluid() wären Anfangs-Werte die aus tcl übergeben werden*/
    /* default values for fields in lattice units */
    gpu_check[0] = 1;

    float Rho = para.rho*para.agrid*para.agrid*para.agrid;
    float v[3] = { 0.0f, 0.0f, 0.0f };
    float pi[6] = { Rho*c_sound_sq, 0.0f, Rho*c_sound_sq, 0.0f, 0.0f, Rho*c_sound_sq };

    float rhoc_sq = Rho*c_sound_sq;
    float avg_rho = para.rho*para.agrid*para.agrid*para.agrid;
    float local_rho, local_j[3], *local_pi, trace;

    local_rho  = Rho;

    local_j[0] = Rho * v[0];
    local_j[1] = Rho * v[1];
    local_j[2] = Rho * v[2];

    local_pi = pi;

    /* reduce the pressure tensor to the part needed here */
    local_pi[0] -= rhoc_sq;
    local_pi[2] -= rhoc_sq;
    local_pi[5] -= rhoc_sq;

    trace = local_pi[0] + local_pi[2] + local_pi[5];

    float rho_times_coeff;
    float tmp1,tmp2;

    /* update the q=0 sublattice */
    n_a.vd[0][index] = 1.f/3.f * (local_rho-avg_rho) - 1.f/2.f*trace;

    /* update the q=1 sublattice */
    rho_times_coeff = 1.f/18.f * (local_rho-avg_rho);

    n_a.vd[1][index] = rho_times_coeff + 1.f/6.f*local_j[0] + 1.f/4.f*local_pi[0] - 1.f/12.f*trace;
    n_a.vd[2][index] = rho_times_coeff - 1.f/6.f*local_j[0] + 1.f/4.f*local_pi[0] - 1.f/12.f*trace;
    n_a.vd[3][index] = rho_times_coeff + 1.f/6.f*local_j[1] + 1.f/4.f*local_pi[2] - 1.f/12.f*trace;
    n_a.vd[4][index] = rho_times_coeff - 1.f/6.f*local_j[1] + 1.f/4.f*local_pi[2] - 1.f/12.f*trace;
    n_a.vd[5][index] = rho_times_coeff + 1.f/6.f*local_j[2] + 1.f/4.f*local_pi[5] - 1.f/12.f*trace;
    n_a.vd[6][index] = rho_times_coeff - 1.f/6.f*local_j[2] + 1.f/4.f*local_pi[5] - 1.f/12.f*trace;

    /* update the q=2 sublattice */
    rho_times_coeff = 1.f/36.f * (local_rho-avg_rho);

    tmp1 = local_pi[0] + local_pi[2];
    tmp2 = 2.0f*local_pi[1];
    n_a.vd[7][index]  = rho_times_coeff + 1.f/12.f*(local_j[0]+local_j[1]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[8][index]  = rho_times_coeff - 1.f/12.f*(local_j[0]+local_j[1]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[9][index]  = rho_times_coeff + 1.f/12.f*(local_j[0]-local_j[1]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
    n_a.vd[10][index] = rho_times_coeff - 1.f/12.f*(local_j[0]-local_j[1]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;

    tmp1 = local_pi[0] + local_pi[5];
    tmp2 = 2.0f*local_pi[3];

    n_a.vd[11][index] = rho_times_coeff + 1.f/12.f*(local_j[0]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[12][index] = rho_times_coeff - 1.f/12.f*(local_j[0]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[13][index] = rho_times_coeff + 1.f/12.f*(local_j[0]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
    n_a.vd[14][index] = rho_times_coeff - 1.f/12.f*(local_j[0]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;

    tmp1 = local_pi[2] + local_pi[5];
    tmp2 = 2.0f*local_pi[4];

    n_a.vd[15][index] = rho_times_coeff + 1.f/12.f*(local_j[1]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[16][index] = rho_times_coeff - 1.f/12.f*(local_j[1]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[17][index] = rho_times_coeff + 1.f/12.f*(local_j[1]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
    n_a.vd[18][index] = rho_times_coeff - 1.f/12.f*(local_j[1]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;

    /*set different seed for randomgen on every node */
    n_a.seed[index] = para.your_seed + index;
  }
}
/*-------------------------------------------------------*/
/** kernel for the initalisation of the particle force array*/
/*@{
 * @param *particle_force	Pointer to local particle force (Output)
 * @param *part				Pointer to the particle rn seed storearray (Output)
}*/
/*-------------------------------------------------------*/
__global__ void init_particle_force(LB_particle_force_gpu *particle_force, LB_particle_seed_gpu *part){
	
  unsigned int part_index = blockDim.x * blockIdx.x + threadIdx.x;
	
  if(part_index<para.number_of_particles){
    particle_force[part_index].f[0] = 0.0f;
    particle_force[part_index].f[1] = 0.0f;
    particle_force[part_index].f[2] = 0.0f;
	
    part[part_index].seed = para.your_seed + part_index;
  }
			
}
/*-------------------------------------------------------*/
/** kernel for the initalisation of the partikel force array */
/*@{
 * @param *particle_force	pointer to local particle force (Input)
}*/
/*-------------------------------------------------------*/
__global__ void reset_particle_force(LB_particle_force_gpu *particle_force){
	
  unsigned int part_index = blockDim.x * blockIdx.x + threadIdx.x;
	
  if(part_index<para.number_of_particles){
    particle_force[part_index].f[0] = 0.0f;
    particle_force[part_index].f[1] = 0.0f;
    particle_force[part_index].f[2] = 0.0f;
  }			
}
/*-------------------------------------------------------*/
/** (re-)initialization of the node force / set up of external force in lb units */
/*@{
 * @param node_f		Pointer to local node force (Input)
}*/
/*-------------------------------------------------------*/
__global__ void reinit_node_force(LB_node_force_gpu node_f){

  unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    if(para.external_force){
      node_f.force[0][index] = para.ext_force[0]*para.agrid*para.agrid*para.tau*para.tau;
      node_f.force[1][index] = para.ext_force[1]*para.agrid*para.agrid*para.tau*para.tau;
      node_f.force[2][index] = para.ext_force[2]*para.agrid*para.agrid*para.tau*para.tau;
    }
    else{
      node_f.force[0][index] = 0.0f;
      node_f.force[1][index] = 0.0f;
      node_f.force[2][index] = 0.0f;
    }
  }
}
#if 1
/*-------------------------------------------------------*/
/**hard coded boundary kernel for custom made boundaries */
/**just for advanced LB users to setup special boundaries or mark some nodes with
	the boundary flag e.g. to reset this nodes*/
/*@{
 * @param n_a		Pointer to local node residing in array a (Input/Output)
 * @param n_b		Pointer to local node residing in array b (Input/Output)
}*/
/*-------------------------------------------------------*/
__global__ void init_boundaries_hardcoded(LB_nodes_gpu n_a, LB_nodes_gpu n_b){
	
  unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){	
#if 1
    unsigned int xyz[3];
    index_to_xyz(index, xyz);
    unsigned int x = xyz[0];
#endif
#if 0
    unsigned int y = xyz[1];
    unsigned int z = xyz[2];

    n_a.boundary[index] = n_b.boundary[index] = 0;
	
    /* bottomplate || topplate*/	
    if(index < para.dim_x*para.dim_y || index >= (para.dim_z-1)*para.dim_x*para.dim_y){
      n_a.boundary[index] = n_b.boundary[index] = 1;	
    }
#endif
#if 0
    if(x == (para.dim_x/8) && y > ((35*para.dim_y)/100)  && y < ((65*para.dim_y)/100) && z > (25*para.dim_z/100) && z < (75*para.dim_z/100)){
      n_a.boundary[index] = n_b.boundary[index] = 1;
    }
#endif	
#if 1
    if(x == 0){
      n_a.boundary[index] = n_b.boundary[index] = 2;
    }
#endif	
  }
}
#endif
/*-------------------------------------------------------*/
/**set the boundary flag for all boundary nodes */
/*@{
 * @param *boundindex	Pointer to the 1d index of the boundnode (Input)
 * @param n_a			Pointer to local node residing in array a (Input)
 * @param n_b			Pointer to local node residing in array b (Input)
}*/
/*-------------------------------------------------------*/
__global__ void init_boundaries(int *boundindex, int number_of_boundnodes, LB_nodes_gpu n_a, LB_nodes_gpu n_b){

  unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;

  if(index<number_of_boundnodes){
    n_a.boundary[boundindex[index]] = n_b.boundary[boundindex[index]] = 1;
  }	
}
/*-------------------------------------------------------*/
/**reset the boundary flag of every node */
/*@{
 * @param n_a		Pointer to local node residing in array a (Input)
 * @param n_b		Pointer to local node residing in array b (Input)	
}*/
/*-------------------------------------------------------*/
__global__ void reset_boundaries(LB_nodes_gpu n_a, LB_nodes_gpu n_b){

  unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    n_a.boundary[index] = n_b.boundary[index] = 0;
  }
}
/*-------------------------------------------------------*/
/** integrationstep of the lb-fluid-solver */
/*@{
 * @param n_a		Pointer to local node residing in array a (Input)
 * @param n_b		Pointer to local node residing in array b (Input)
 * @param *d_v		Pointer to local device values (Input)
 * @param node_f	Pointer to local node force (Input)
}*/
/*-------------------------------------------------------*/
__global__ void integrate(LB_nodes_gpu n_a, LB_nodes_gpu n_b, LB_values_gpu *d_v, LB_node_force_gpu node_f){
    
  /**every node is connected to a thread via the index*/
  unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
  /**the 19 moments (modes) are only temporary register values */
  float mode[19];
  LB_randomnr_gpu rng;

  if(index<para.number_of_nodes){
    /** storing the seed into a register value*/
    rng.seed = n_a.seed[index];
    /**calc_m_from_n*/
    calc_m_from_n(n_a, index, mode);
    /**lb_relax_modes*/
    relax_modes(mode, index, node_f);
    /**lb_thermalize_modes */
    if (para.fluct) thermalize_modes(mode, index, &rng);
#ifdef EXTERNAL_FORCES
    /**if external force is used apply node force */
    if (para.external_force) apply_forces(index, mode, node_f);
#else
    /**if partcles are used apply node forces*/
    if (para.number_of_particles) apply_forces(index, mode, node_f); 
#endif
    /**lb_calc_n_from_modes_push*/
    normalize_modes(mode);
    /**calc of velocity densities and streaming with pbc*/
    calc_n_from_modes_push(n_b, mode, index);
    /** rewriting the seed back to the global memory*/
    n_b.seed[index] = rng.seed;
  }  
}
/*-------------------------------------------------------*/
/** part interaction kernel */
/*@{
 * @param n_a				Pointer to local node residing in array a (Input)
 * @param *particle_data	Pointer to the particle position and velocity (Input)
 * @param *particle_force	Pointer to the particle force (Input)
 * @param *part				Pointer to the rn array of the particles (Input)
 * @param node_f			Pointer to local node force (Input)
}*/
/*-------------------------------------------------------*/
__global__ void calc_fluid_particle_ia(LB_nodes_gpu n_a, LB_particle_gpu *particle_data, LB_particle_force_gpu *particle_force, LB_node_force_gpu node_f, LB_particle_seed_gpu *part){
	
  unsigned int part_index = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int node_index[8];
  float delta[8];
  float delta_j[3];
  LB_randomnr_gpu rng_part;
	
  if(part_index<para.number_of_particles){
#if 0		
    /** check if particles are in the box*/
    check_part_posis(particle_data, part_index);		
#endif
    rng_part.seed = part[part_index].seed;
    /**calc of the force which act on the particle */
    calc_viscous_force(n_a, delta, particle_data, particle_force, part_index, &rng_part, delta_j, node_index);
    /**calc of the force which acts back to the fluid node */
    calc_node_force(delta, delta_j, node_index, node_f);
    part[part_index].seed = rng_part.seed;		
  }
}
/*-------------------------------------------------------*/
/**Bounce back boundary read kernel*/
/*@{
 * @param n_a					Pointer to local node residing in array a (Input)
 * @param n_b					Pointer to local node residing in array b (Input)
}*/
/*-------------------------------------------------------*/
__global__ void bb_read(LB_nodes_gpu n_a, LB_nodes_gpu n_b){

  unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    bounce_back_read(n_b, n_a, index);
  }
}
/*-------------------------------------------------------*/
/**Bounce back boundary write kernel*/
/*@{
 * @param n_a					Pointer to local node residing in array a (Input)
 * @param n_b					Pointer to local node residing in array b (Input)
}*/
/*-------------------------------------------------------*/
__global__ void bb_write(LB_nodes_gpu n_a, LB_nodes_gpu n_b){

  unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    bounce_back_write(n_b, n_a, index);
  }
}
/*-------------------------------------------------------*/
/*@{
 * @param n_a		Pointer to local node residing in array a (Input)
 * @param n_b		Pointer to local node residing in array b (Input)
}*/
/*-------------------------------------------------------*/
__global__ void reset_population(LB_nodes_gpu n_b, LB_nodes_gpu n_a){
 	
  unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    reset_pop(n_b, n_a, index);
  }
}
/*-------------------------------------------------------*/
/*@{
 * @param n_a		Pointer to local node residing in array a (Input)
 * @param *d_v		Pointer to local device values (Input)
}*/
/*-------------------------------------------------------*/
__global__ void values(LB_nodes_gpu n_a, LB_values_gpu *d_v){

  float mode[19];
  unsigned int singlenode = 0;
  unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    calc_mode(mode, n_a, index);
    calc_values(n_a, mode, d_v, index, singlenode);
  }
}

__global__ void init_extern_nodeforces(int n_extern_nodeforces, LB_extern_nodeforce_gpu *extern_nodeforces, LB_node_force_gpu node_f){

  unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;

  if(index<n_extern_nodeforces){
    node_f.force[0][extern_nodeforces[index].index] = extern_nodeforces[index].force[0]*para.agrid*para.agrid*para.tau*para.tau;
    node_f.force[1][extern_nodeforces[index].index] = extern_nodeforces[index].force[1]*para.agrid*para.agrid*para.tau*para.tau;
    node_f.force[2][extern_nodeforces[index].index] = extern_nodeforces[index].force[2]*para.agrid*para.agrid*para.tau*para.tau;
  }
}
__global__ void lb_print_node(int single_nodeindex, LB_values_gpu *d_p_v, LB_nodes_gpu n_a){
	
  float mode[19];
  unsigned int singlenode = 1;

  if(blockDim.x * blockIdx.x + threadIdx.x == 0){
    calc_mode(mode, n_a, single_nodeindex);
    calc_values(n_a, mode, d_p_v, single_nodeindex, singlenode);
  }	
}

void cuda_safe_mem(hipError_t err){
    if( hipSuccess != err) {                                             
      fprintf(stderr, "Could not allocate gpu memory.\n");
      exit(EXIT_FAILURE);
    }
}
void cuda_safe_kernel(hipError_t err){
    if( hipSuccess != err) {                                             
      fprintf(stderr, "cuda kernel failed! (maybe your dimensions are to large).\n");
      exit(EXIT_FAILURE);
    }
}
/**********************************************************************/
/* Host funktions to setup and call kernels*/
/**********************************************************************/
/**-------------------------------------------------------*/
/*@{
 * @param *lbpar_gpu	Pointer to parameters to setup the lb field
}*/
/**-------------------------------------------------------*/
void lb_init_GPU(LB_parameters_gpu *lbpar_gpu){

  // Allocate lattice-struct in device memory
  size_of_values = lbpar_gpu->number_of_nodes * sizeof(LB_values_gpu);
  size_of_forces = lbpar_gpu->number_of_particles * sizeof(LB_particle_force_gpu);
  size_of_positions = lbpar_gpu->number_of_particles * sizeof(LB_particle_gpu);
  size_of_seed = lbpar_gpu->number_of_particles * sizeof(LB_particle_seed_gpu);

  cuda_safe_mem(hipMalloc((void**)&device_values, size_of_values));

  for(int i=0; i<19; i++){
    cuda_safe_mem(hipMalloc((void**)&nodes_a.vd[i], lbpar_gpu->number_of_nodes * sizeof(float)));
    cuda_safe_mem(hipMalloc((void**)&nodes_b.vd[i], lbpar_gpu->number_of_nodes * sizeof(float)));                                           
  }
  cuda_safe_mem(hipMalloc((void**)&nodes_a.seed, lbpar_gpu->number_of_nodes * sizeof(unsigned int)));
  cuda_safe_mem(hipMalloc((void**)&nodes_a.boundary, lbpar_gpu->number_of_nodes * sizeof(unsigned int)));
  cuda_safe_mem(hipMalloc((void**)&nodes_b.seed, lbpar_gpu->number_of_nodes * sizeof(unsigned int)));
  cuda_safe_mem(hipMalloc((void**)&nodes_b.boundary, lbpar_gpu->number_of_nodes * sizeof(unsigned int)));

  for(int i=0; i<3; i++){
    cuda_safe_mem(hipMalloc((void**)&node_f.force[i], lbpar_gpu->number_of_nodes * sizeof(float)));
  }
  
  cuda_safe_mem(hipMalloc((void**)&particle_force, size_of_forces));
  cuda_safe_mem(hipMalloc((void**)&particle_data, size_of_positions));	
  cuda_safe_mem(hipMalloc((void**)&part, size_of_seed));
	
  /**write parameters in const memory*/
  hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu));
  /**check flag if lb gpu init works*/
  cuda_safe_mem(hipMalloc((void**)&gpu_check, sizeof(int)));
  h_gpu_check = (int*)malloc(sizeof(int));

  /** values for the kernel call */
  if((lbpar_gpu->dim_x*lbpar_gpu->dim_y*lbpar_gpu->dim_z/64) > 65535){
    threads_per_block = 128;
  }else
    threads_per_block = 64;

  blocks_per_grid = (lbpar_gpu->number_of_nodes + threads_per_block - 1) /(threads_per_block);

  /** values for the particle kernel */
  threads_per_block_particles = 64;
  blocks_per_grid_particles = (lbpar_gpu->number_of_particles + threads_per_block_particles - 1)/(threads_per_block_particles);

  reset_boundaries<<<blocks_per_grid, threads_per_block>>>(nodes_a, nodes_b);
  cuda_safe_kernel(hipGetLastError());	
  /** calc of veloctiydensities from given parameters and initialize the Node_Force array with zero */
  calc_n_equilibrium<<<blocks_per_grid, threads_per_block>>>(nodes_a, node_f, gpu_check);
  cuda_safe_kernel(hipGetLastError());	
  /** init part forces with zero*/
  if(lbpar_gpu->number_of_particles) init_particle_force<<<blocks_per_grid_particles, threads_per_block_particles>>>(particle_force, part);
  cuda_safe_kernel(hipGetLastError());	
  reinit_node_force<<<blocks_per_grid, threads_per_block>>>(node_f);
  cuda_safe_kernel(hipGetLastError());	

  hipStreamCreate(&stream[0]);

  h_gpu_check[0] = 0;
  hipMemcpy(h_gpu_check, gpu_check, sizeof(int), hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

  if(!h_gpu_check[0]){
    fprintf(stderr, "initialization of lb gpu code failed! \n");
    errexit();	
  }	
}
/**-------------------------------------------------------------------------*/
/**setup and call particle reallocation from the host */
/*@{
 * @param *lbpar_gpu	Pointer to parameters to setup the lb field
}*/
/**-------------------------------------------------------------------------*/
void lb_realloc_particle_GPU(LB_parameters_gpu *lbpar_gpu){

  hipFree(particle_force);
  hipFree(particle_data);
  hipFree(part);

  hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu));

  size_of_forces = lbpar_gpu->number_of_particles * sizeof(LB_particle_force_gpu);
  size_of_positions = lbpar_gpu->number_of_particles * sizeof(LB_particle_gpu);
  size_of_seed = lbpar_gpu->number_of_particles * sizeof(LB_particle_seed_gpu);

  cuda_safe_mem(hipMalloc((void**)&particle_force, size_of_forces));
  cuda_safe_mem(hipMalloc((void**)&particle_data, size_of_positions));
  cuda_safe_mem(hipMalloc((void**)&part, size_of_seed));

  /** values for the particle kernel */
  threads_per_block_particles = 64;
  blocks_per_grid_particles = (lbpar_gpu->number_of_particles + threads_per_block_particles - 1)/(threads_per_block_particles);

  if(lbpar_gpu->number_of_particles) init_particle_force<<<blocks_per_grid_particles, threads_per_block_particles>>>(particle_force, part);
	
  if(lbpar_gpu->number_of_particles) reinit_node_force<<<blocks_per_grid, threads_per_block>>>(node_f);
	
}

/**-------------------------------------------------------------------------*/
/**setup and call boundaries from the host */
/*@{
 * @param *host_boundindex		Pointer to the host bound index
 * @param number_of_boundnodes	number of boundnodes
}*/
/**-------------------------------------------------------------------------*/
void lb_init_boundaries_GPU(int number_of_boundnodes, int *host_boundindex){

  size_of_boundindex = number_of_boundnodes*sizeof(int);
  hipMemcpyToSymbol(HIP_SYMBOL(number_of_bnodes), &number_of_boundnodes, sizeof(int));
  hipMalloc((void**)&boundindex, size_of_boundindex);
  hipMemcpy(boundindex, host_boundindex, size_of_boundindex, hipMemcpyHostToDevice);

  reset_boundaries<<<blocks_per_grid, threads_per_block>>>(nodes_a, nodes_b);
  cuda_safe_kernel(hipGetLastError());	

  if((lbpar_gpu.dim_x*lbpar_gpu.dim_y*lbpar_gpu.dim_z/64) > 65535){
    threads_per_block_bound = 128;
  }else
    threads_per_block_bound = 64;
  blocks_per_grid_bound = (number_of_boundnodes + threads_per_block_bound -1)/(threads_per_block_bound);

#if 0
  init_boundaries_hardcoded<<<blocks_per_grid_bound, threads_per_block_bound>>>(nodes_a, nodes_b);
#endif
  init_boundaries<<<blocks_per_grid_bound, threads_per_block_bound>>>(boundindex, number_of_boundnodes, nodes_a, nodes_b);
  cuda_safe_kernel(hipGetLastError());	
  calc_n_equilibrium<<<blocks_per_grid, threads_per_block>>>(nodes_a, node_f, gpu_check);
  cuda_safe_kernel(hipGetLastError());	
  hipDeviceSynchronize();
}
void lb_init_extern_nodeforces_GPU(int n_extern_nodeforces, LB_extern_nodeforce_gpu *host_extern_nodeforces, LB_parameters_gpu *lbpar_gpu){

  size_of_extern_nodeforces = n_extern_nodeforces*sizeof(LB_extern_nodeforce_gpu);
  hipMalloc((void**)&extern_nodeforces, size_of_extern_nodeforces);
  hipMemcpy(extern_nodeforces, host_extern_nodeforces, size_of_extern_nodeforces, hipMemcpyHostToDevice);

  if(para.external_force == 0)hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu)); 

  threads_per_block_exf = 128;

  blocks_per_grid_exf = (n_extern_nodeforces + threads_per_block_exf -1)/(threads_per_block_exf);
	
  init_extern_nodeforces<<<blocks_per_grid_exf, threads_per_block_exf>>>(n_extern_nodeforces, extern_nodeforces, node_f);
  cuda_safe_kernel(hipGetLastError());	
}

/**-------------------------------------------------------------------------*/
/**setup and call particle kernel from the host */
/*@{
 * @param **host_data		Pointer to the host particle positions and velocities
}*/
/**-------------------------------------------------------------------------*/
void lb_particle_GPU(LB_particle_gpu *host_data){
  	
  /** get espresso md particle values*/
  hipMemcpyAsync(particle_data, host_data, size_of_positions, hipMemcpyHostToDevice, stream[0]);

  /** call of the particle kernel */
  calc_fluid_particle_ia<<<blocks_per_grid_particles, threads_per_block_particles, 0, stream[0]>>>(nodes_a, particle_data, particle_force, node_f, part);
  cuda_safe_kernel(hipGetLastError());	
}
/** setup and call kernel to copy particle forces to host */
void lb_copy_forces_GPU(LB_particle_force_gpu *host_forces){

  /** Copy result from device memory to host memory*/
  hipMemcpy(host_forces, particle_force, size_of_forces, hipMemcpyDeviceToHost);

  /** reset part forces with zero*/
  reset_particle_force<<<blocks_per_grid_particles, threads_per_block_particles, 0,  stream[0]>>>(particle_force);
  cuda_safe_kernel(hipGetLastError());	
  hipDeviceSynchronize();
}

/** setup and call kernel for getting macroscopic fluid values of all nodes*/
void lb_get_values_GPU(LB_values_gpu *host_values){

  values<<<blocks_per_grid, threads_per_block>>>(nodes_a, device_values);
  cuda_safe_kernel(hipGetLastError());	

  hipMemcpy(host_values, device_values, size_of_values, hipMemcpyDeviceToHost);

  hipDeviceSynchronize();

}
/** setup and call kernel for getting macroscopic fluid values of a single node*/
void lb_print_node_GPU(int single_nodeindex, LB_values_gpu *host_print_values){ 
      
  LB_values_gpu *device_print_values;
  hipMalloc((void**)&device_print_values, sizeof(LB_values_gpu));	
  threads_per_block_print = 1;
  blocks_per_grid_print = 1;
  lb_print_node<<<blocks_per_grid_print, threads_per_block_print>>>(single_nodeindex, device_print_values, nodes_a);
  hipMemcpy(host_print_values, device_print_values, sizeof(LB_values_gpu), hipMemcpyDeviceToHost);
  cuda_safe_kernel(hipGetLastError());

  hipDeviceSynchronize();
}

/**-------------------------------------------------------------------------*/
			/**setup and call integrate kernel from the host */
/**-------------------------------------------------------------------------*/
void lb_integrate_GPU(){
		
  /**call of fluid step*/
  if (intflag == 1){
    (integrate<<<blocks_per_grid, threads_per_block, 0,  stream[0]>>>(nodes_a, nodes_b, device_values, node_f));
    cuda_safe_kernel(hipGetLastError());
#if 0		
    reset_population<<<blocks_per_grid, threads_per_block, 0,  stream[0]>>>(nodes_b, nodes_a);
    cuda_safe_kernel(hipGetLastError());
#endif
#ifdef LB_BOUNDARIES_GPU		
    if (lb_boundaries_bb_gpu == 1) bb_read<<<blocks_per_grid, threads_per_block, 0,  stream[0]>>>(nodes_a, nodes_b);
      cuda_safe_kernel(hipGetLastError());			
    if (lb_boundaries_bb_gpu == 1) bb_write<<<blocks_per_grid, threads_per_block, 0,  stream[0]>>>(nodes_a, nodes_b);
      cuda_safe_kernel(hipGetLastError());
#endif
    intflag = 0;
  }
  else{
    integrate<<<blocks_per_grid, threads_per_block, 0,  stream[0]>>>(nodes_b, nodes_a, device_values, node_f);
    cuda_safe_kernel(hipGetLastError());
#if 0		
    reset_population<<<blocks_per_grid, threads_per_block, 0,  stream[0]>>>(nodes_a, nodes_b);
    cuda_safe_kernel(hipGetLastError());
#endif
#ifdef LB_BOUNDARIES_GPU		
    if (lb_boundaries_bb_gpu == 1) bb_read<<<blocks_per_grid, threads_per_block, 0,  stream[0]>>>(nodes_b, nodes_a);
      cuda_safe_kernel(hipGetLastError());
			
    if (lb_boundaries_bb_gpu == 1) bb_write<<<blocks_per_grid, threads_per_block, 0,  stream[0]>>>(nodes_b, nodes_a);
      cuda_safe_kernel(hipGetLastError());
#endif
    intflag = 1;
  }             
}

/**-------------------------------------------------------------------------*/
			/** free gpu memory kernel called from the host */
/**-------------------------------------------------------------------------*/
void lb_free_GPU(){
  // Free device memory
  hipFree(device_values);
  hipFree(&para);
  hipFree(&nodes_a);
  hipFree(&nodes_b);
  hipFree(particle_force);
  hipFree(particle_data);
  hipFree(&node_f);
  hipFree(part);
  hipFree(&number_of_bnodes);
  hipStreamDestroy(stream[0]);
}
#endif /* LB_GPU */
